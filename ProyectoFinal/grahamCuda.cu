#include "hip/hip_runtime.h"
// A C++ program to find convex hull of a set of points. Refer
// http://www.geeksforgeeks.org/orientation-3-ordered-points/
// for explanation of orientation()

#include <iostream>
#include <stack>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "mergeCuda.cu"

using namespace std;

struct Point
{
    int x, y;
};

// A globle point needed for  sorting points with reference
// to  the first point Used in compare function of qsort()
Point p0;

// A utility function to find next to top in a stack
Point nextToTop(stack<Point> &S)
{
    Point p = S.top();
    S.pop();
    Point res = S.top();
    S.push(p);
    return res;
}

// A utility function to swap two points
void swap(Point &p1, Point &p2)
{
    Point temp = p1;
    p1 = p2;
    p2 = temp;
}

// A utility function to return square of distance
// between p1 and p2
int distSq(Point p1, Point p2)
{
    return (p1.x - p2.x)*(p1.x - p2.x) +
          (p1.y - p2.y)*(p1.y - p2.y);
}

// To find orientation of ordered triplet (p, q, r).
// The function returns following values
// 0 --> p, q and r are colinear
// 1 --> Clockwise
// 2 --> Counterclockwise
int orientation(Point p, Point q, Point r)
{
    int val = (q.y - p.y) * (r.x - q.x) -
              (q.x - p.x) * (r.y - q.y);

    if (val == 0) return 0;  // colinear
    return (val > 0)? 1: 2; // clock or counterclock wise
}

// A function used by library function qsort() to sort an array of
// points with respect to the first point
int compare(const void *vp1, const void *vp2)
{
   Point *p1 = (Point *)vp1;
   Point *p2 = (Point *)vp2;

   // Find orientation
   int o = orientation(p0, *p1, *p2);
   if (o == 0)
     return (distSq(p0, *p2) >= distSq(p0, *p1))? -1 : 1;

   return (o == 2)? -1: 1;
}

void mainMergeSortCuda(long *v, long n){

    dim3 threadsPerBlock;
    dim3 blocksPerGrid;

    threadsPerBlock.x = 32;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;

    blocksPerGrid.x = 8;
    blocksPerGrid.y = 1;
    blocksPerGrid.z = 1;

    //
    // Read numbers from stdin
    //
    long* data;
    long size = n;
    data = v;
    
    for(int i = 0; i < n; i++){
      cout<<data[i]<<endl;
    }

    // merge-sort the data
    //mergesort(data, size, threadsPerBlock, blocksPerGrid);

    //
    // Print out the list
    //
    for (int i = 0; i < size; i++) {
        std::cout << data[i] << '\n';
    }
}

// Prints convex hull of a set of n points.
void convexHull(Point points[], int n)
{
   // Find the bottommost point
   int ymin = points[0].y, min = 0;
   for (int i = 1; i < n; i++)
   {
     int y = points[i].y;

     // Pick the bottom-most or chose the left
     // most point in case of tie
     if ((y < ymin) || (ymin == y &&
         points[i].x < points[min].x))
        ymin = points[i].y, min = i;
   }

   // Place the bottom-most point at first position
   swap(points[0], points[min]);

   // Sort n-1 points with respect to the first point.
   // A point p1 comes before p2 in sorted ouput if p2
   // has larger polar angle (in counterclockwise
   // direction) than p1
   p0 = points[0];
   cout<<points[0].x<<" "<<points[0].y<<endl;

   //qsort(&points[1], n-1, sizeof(Point), compare);

   cout << "sacando distancias"<<endl; 
   long *distance;
   distance = (long*)malloc(n-1*sizeof(long));

   //extract distances
   for(int i = 0; i < n-1; i++){
    distance[i] = distSq(p0,points[i+1]);
    cout<<"("<<distance[i]<<")"<<endl;
   }

   long size = n-1;
   dim3 dimBlock(32,1,1);
   dim3 dimGrid(8,1,1);

   cout<<"ordenando distancias"<<endl;

   mainMergeSortCuda(distance, size);

   for(int i = 0; i < n-1; i++){
      cout<<"["<<distance[i]<<"]"<<endl;
   }

   cout<<"datos ordenados"<<endl;

   // If two or more points make same angle with p0,
   // Remove all but the one that is farthest from p0
   // Remember that, in above sorting, our criteria was
   // to keep the farthest point at the end when more than
   // one points have same angle.
   int m = 1; // Initialize size of modified array
   for (int i=1; i<n; i++)
   {
       // Keep removing i while angle of i and i+1 is same
       // with respect to p0
       while (i < n-1 && orientation(p0, points[i],
                                    points[i+1]) == 0)
          i++;


       points[m] = points[i];
       m++;  // Update size of modified array
   }

   // If modified array of points has less than 3 points,
   // convex hull is not possible
   if (m < 3) return;

   // Create an empty stack and push first three points
   // to it.
   stack<Point> S;
   S.push(points[0]);
   S.push(points[1]);
   S.push(points[2]);

   // Process remaining n-3 points
   for (int i = 3; i < m; i++)
   {
      // Keep removing top while the angle formed by
      // points next-to-top, top, and points[i] makes
      // a non-left turn
      while (orientation(nextToTop(S), S.top(), points[i]) != 2)
         S.pop();
      S.push(points[i]);
   }

   cout << S.size() << endl;

   // Now stack has the output points, print contents of stack
   while (!S.empty())
   {
       Point p = S.top();
       cout << p.x << " " << p.y << endl;
       S.pop();
   }
   free(distance);
}

// Driver program to test above functions
int main(){
    int n;
    cin >> n;
    cout << n << endl;
    Point points[n];

    for(int i = 0;  i < n; i++){
        cin >> points[i].x;
        cin >> points[i].y;
        cout << points[i].x << " " << points[i].y << endl;
    }

    convexHull(points, n);
    return 0;
}
