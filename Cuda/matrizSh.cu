#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<malloc.h>
#include<time.h>
#include<hip/hip_runtime.h>

__global__
void matrixMultKernel(float* d_M, float* d_N, float* d_P, int width){
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    float Pvalue = 0;

    for(int i = 0; i < width/TILE_WIDTH; ++i){
      Mds[ty][tx] = d_M[Row*width + m*TILE_WIDTH + tx];
      Nds[ty][tx] = d_N[(m*TILE_WIDTH + ty)*width + Col];
      __syncthreads();

      for(int k=0; k<TILE_WIDTH; ++k){
        Pvalue += Mds[ty][k] + Nds[k][tx];
      }
      __syncthreads();
    }
    d_P[Row*width + Col] = Pvalue;
}


__host__
void print(float *M, int size){
    printf("-----------Vector------------\n");
    for(int i=0; i<size; i++){
        printf("%f", M[i]);
        printf("\n");
    }
}

__host__
void receive(float *M, FILE *stream, int size){
    for(int i=0; i<size; i++){
        fscanf(stream, "%f", &M[i]);
    }
    fclose(stream);
}

int main(int argc, char** argv){
    if(argc != 3){
        printf("Must be called with the names of the files \n");
        return 1;
    }

    int sizeA, sizeB;

    hipError_t error = hipSuccess;
    float *h_A, *h_B, *h_C;
    FILE *f1, *f2;
    f1 = fopen(argv[1], "r");
    f2 = fopen(argv[2], "r");

    fscanf(f1, "%d", &sizeA);
    fscanf(f2, "%d", &sizeB);

    if(sizeA != sizeB){
        printf("The vectors should have same dimensions \b");
        return 1;
    }

    //CPU
    h_A = (float*)malloc(sizeA*sizeof(float));
    h_B = (float*)malloc(sizeA*sizeof(float));
    h_C = (float*)malloc(sizeA*sizeof(float));

    receive(h_A, f1, sizeA);
    receive(h_B, f2, sizeA);
    //print(h_A, sizeA);
    //print(h_B, sizeB);

    //GPU
    float *d_A, *d_B, *d_C;
    int blockSize = 32;
    int gridSize = ceil(sizeA / float(blockSize));

    //dim3 dimBlock(blockSize,1,1);
    //dim3 dimGrid(ceil(sizeA / float(blockSize)),1,1);

    error = hipMalloc((void**)&d_A, sizeA*sizeof(float));
    if (error != hipSuccess){
        printf("Error allocating memory d_A");
        return 1;
    }

    error = hipMalloc((void**)&d_B, sizeA*sizeof(float));
    if (error != hipSuccess){
        printf("Error allocating memory d_B");
        return 1;
    }

    error = hipMalloc((void**)&d_C, sizeA*sizeof(float));
    if (error != hipSuccess){
        printf("Error allocating memory d_C");
        return 1;
    }

    hipMemcpy(d_A, h_A, sizeA*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeA*sizeof(float), hipMemcpyHostToDevice);

    sum<<<gridSize, blockSize>>>(d_A, d_B, d_C, sizeA);
    //hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, sizeA*sizeof(float), hipMemcpyDeviceToHost);
    print(h_C, sizeA);

    free(h_A); free(h_B); free(h_C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    return 0;
}
