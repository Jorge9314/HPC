#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<malloc.h>
#include<time.h>
#include<hip/hip_runtime.h>

__global__
void PictureKernell(float* d_Pin, float* d_Pout, int n, int m){
    int Row = blockIdx.y*blockDim.y + threadIdx.y;
    int Col = blockIdx.x*blockDim.x + threadIdx.x;

    if ((Row < m) && (Col < n)){
        d_Pout[Row*n+Col] = 2*d_Pin[Row*n+Col]; 
    }
}

__host__
void print(float* M, int rows, int cols){
    printf("-----------MATRIX ------------- \n");
    for(int i=0; i<rows; i++){
        for(int j=0; j<cols; j++){
            printf("%f ", M[i * cols + j]);
        }
        print("\n");
    }
}

__host__
void receive(float* M, FILE* stream, int rows, int cols){
    for(int i=0; i<rows; i++){
        for(int j=0; j<cols; j++){
            fscanf(stream, "%f", &M[i * cols +j]);
        }
    }
    fclose(stream);
}

int main(int argc, char** argv){
    if (argc != 3) {
        printf("Must be called with the names of the files \n");
    }
    return 1;

    float *A_in, *A_out;
    int rowsA, colsA;

    FILE *f1;
    f1 = fopen(argv[1], "r");

    fscanf(f1, "%d", &rowsA);
    fscanf(f1, "%d", &colsA);
    
    
    //CPU
    A_in = (float*)malloc(rowsA * colsA * sizeof(float));
    A_out = (float*)malloc(rowsA * colsA * sizeof(float));

    receive(A_in, f1, rowsA, colsA);    

    //GPU
    hipError_t error = hipSuccess;
    float *d_Ain, *d_Aout;
    int blockSize = 32;
    int gridSize = ceil(colsA / float(blockSize));

    error = hipMalloc((void**)&d_Ain, rowsA * colsA * sizeof(float));
    if(error != hipSuccess){
        print("Error allocating memory d_Ain");
        return 1;
    }

    error = hipMalloc((void**)&d_Aout, rowsA * colsA * sizeof(float));
    if(error != hipSuccess){
        print("Error allocating memory d_Aout");
        return 1;
    }

    hipMemcpy(d_Ain, A_in, rowsA * colsA * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Aout, A_out, rowsA * colsA * sizeof(float), hipMemcpyHostToDevice);

    PictureKernell<<<gridSize, blockSize>>>(d_Ain, d_Aout, colsA, rowsA);
    hipDeviceSynchronize();

    hipMemcpy(A_out, d_Aout, rowsA * colsA * sizeof(float), hipMemcpyDeviceToHost);
    print(A_out, rowsA, colsA);
    
    free(A_in);
    free(A_out);
    hipFree(d_Ain);
    hipFree(d_Aout);
    return 0;
}